#include "hip/hip_runtime.h"
#include "UpScale.h"

__global__ void Nearest_Neighborhood_Kernel(PIXEL* dLR, PIXEL *dRI, int Sampling, int row, int col, int ROW, int COL, int channels) {
	int RIx = blockDim.x * blockIdx.x + threadIdx.x;
	int RIy = blockDim.y * blockIdx.y + threadIdx.y;
	int c = blockDim.z * blockIdx.z + threadIdx.z;

	int LRx = floor((float)RIx / Sampling + 0.5);
	int LRy = floor((float)RIy / Sampling + 0.5);

	if (RIy < ROW && RIx < COL) {
		dRI[RIy * (COL * channels) + RIx * (channels) + c] = dLR[LRy * (col * channels) + LRx * (channels) + c];
	}
}

__global__ void Bilinear_Kernel(PIXEL* dLR, PIXEL *dRI, int Sampling, int row, int col, int ROW, int COL, int channels) {
	int RIx = blockDim.x * blockIdx.x + threadIdx.x;
	int RIy = blockDim.y * blockIdx.y + threadIdx.y;
	int c = blockDim.z * blockIdx.z + threadIdx.z;

	int LRx = RIx / Sampling;
	int LRy = RIy / Sampling;

	if (RIy < ROW && RIx < COL) {
		int m = RIy % Sampling;
		int n = RIx % Sampling;
		if (m == 0 && n == 0) {
			dRI[RIy * (COL * channels) + RIx * (channels) + c] = dLR[LRy * (col * channels) + LRx * (channels) + c];
		}
		else {
			if (RIy == ROW - 1) {
				LRy--;
				m = Sampling - m;
			}
			if (RIx == COL - 1) {
				LRx--;
				n = Sampling - n;
			}

			PIXEL v1 = dLR[(LRy + 0) * (col * channels) + (LRx + 0) * (channels) + c];
			PIXEL v2 = dLR[(LRy + 0) * (col * channels) + (LRx + 1) * (channels) + c];
			PIXEL v3 = dLR[(LRy + 1) * (col * channels) + (LRx + 0) * (channels) + c];
			PIXEL v4 = dLR[(LRy + 1) * (col * channels) + (LRx + 1) * (channels) + c];

			double w1 = (double)((Sampling - m) * (Sampling - n)) / (Sampling * Sampling);
			double w2 = (double)((Sampling - m) * (n)) / (Sampling * Sampling);
			double w3 = (double)((m) * (Sampling - n)) / (Sampling * Sampling);
			double w4 = (double)((m) * (n)) / (Sampling * Sampling);
			dRI[RIy * (COL * channels) + RIx * (channels) + c] = ((v1 * w1) + (v2 * w2) + (v3 * w3) + (v4 * w4));
		}
	}
}

__device__ __host__ double clamp(const double value, const double min, const double max) {
	return value < min ? min : max < value ? max : value;
}

__device__ __host__ double cubic(PIXEL v1, PIXEL v2, PIXEL v3, PIXEL v4, double d) {
	double result = ((-v1 + (3 * v2) - (3 * v3) + v4) * pow(d, 3)) +
		(((2 * v1) - (5 * v2) + (4 * v3) - v4) * pow(d, 2)) +
		(v3 - v1) * d +
		(2 * v2);
	result /= 2;

	return clamp(result, 0, 255);
}

__global__ void Bicubic_Kernel_Y(PIXEL* dLR, PIXEL *dRI, int Sampling, int row, int col, int ROW, int COL, int channels) {
	int RIx = blockDim.x * blockIdx.x + threadIdx.x;
	int RIy = blockDim.y * blockIdx.y + threadIdx.y;
	int c = blockDim.z * blockIdx.z + threadIdx.z;

	int LRx = RIx / Sampling;
	int LRy = RIy / Sampling;

	if (RIy < ROW && RIx < COL) {
		int m = RIy % Sampling;
		int n = RIx % Sampling;
		if (n != 0) return;

		if (m == 0 && n == 0) {
			dRI[RIy * (COL * channels) + RIx * (channels) + c] = dLR[LRy * (col * channels) + LRx * (channels) + c];
		}
		else {
			double d = (double)m / Sampling;
			if (LRy == 0) {
				LRy += 1; d -= 1;
			}
			else if (LRy == row - 2) {
				LRy -= 1; d += 1;
			}
			PIXEL v1 = dLR[(LRy - 1) * (col * channels) + LRx * (channels) + c];
			PIXEL v2 = dLR[(LRy + 0) * (col * channels) + LRx * (channels) + c];
			PIXEL v3 = dLR[(LRy + 1) * (col * channels) + LRx * (channels) + c];
			PIXEL v4 = dLR[(LRy + 2) * (col * channels) + LRx * (channels) + c];

			dRI[RIy * (COL * channels) + RIx * (channels)+c] = cubic(v1, v2, v3, v4, d);
		}
	}
}

__global__ void Bicubic_Kernel_X(PIXEL* dLR, PIXEL *dRI, int Sampling, int row, int col, int ROW, int COL, int channels) {
	int RIx = blockDim.x * blockIdx.x + threadIdx.x;
	int RIy = blockDim.y * blockIdx.y + threadIdx.y;
	int c = blockDim.z * blockIdx.z + threadIdx.z;

	int LRx = RIx / Sampling;
	int LRy = RIy / Sampling;

	if (RIy < ROW && RIx < COL) {
		int n = RIx % Sampling;
		int tRIx = RIx;
		if (n == 0) return;

		double d = (double)n / Sampling;
		if (tRIx <= Sampling) {
			tRIx += Sampling; d -= 1;
		}
		else if (tRIx >= COL - Sampling) {
			tRIx -= Sampling; d += 1;
		}
		PIXEL v1 = dRI[RIy * (COL * channels) + ((tRIx - n) - (1 * Sampling)) * (channels) + c];
		PIXEL v2 = dRI[RIy * (COL * channels) + ((tRIx - n) + (0 * Sampling)) * (channels) + c];
		PIXEL v3 = dRI[RIy * (COL * channels) + ((tRIx - n) + (1 * Sampling)) * (channels) + c];
		PIXEL v4 = dRI[RIy * (COL * channels) + ((tRIx - n) + (2 * Sampling)) * (channels) + c];

		dRI[RIy * (COL * channels) + RIx * (channels)+c] = cubic(v1, v2, v3, v4, d);
	}
}

Mat GPU_Call(Mat LR, int Sampling, int mod, dim3 gridDim, dim3 blockDim,DS_timer* timer, int timerID) {
	int row = LR.rows;
	int col = LR.cols;
	int channels = LR.channels();

	int ROW = row * Sampling - (Sampling - 1);
	int COL = col * Sampling - (Sampling - 1);

	Mat RI = Mat::zeros(ROW, COL, LR.type());

	int LR_Size = row * col * channels * sizeof(PIXEL);
	int RI_Size = ROW * COL * channels * sizeof(PIXEL);

	PIXEL* dLR = NULL;
	PIXEL* dRI = NULL;
	hipMalloc(&dLR, LR_Size);
	hipMemset(dLR, 0, LR_Size);
	hipMalloc(&dRI, RI_Size);
	hipMemset(dRI, 0, RI_Size);

	hipMemcpy(dLR, LR.data, LR_Size, hipMemcpyHostToDevice);

	timer->onTimer(timerID);
	switch (mod) {
	case Iterpolation::NN:
		Nearest_Neighborhood_Kernel <<<gridDim, blockDim>>>(dLR, dRI, Sampling, row, col, ROW, COL, channels);
		break;
	case Iterpolation::BILINEAR:
		Bilinear_Kernel <<<gridDim, blockDim>>>(dLR, dRI, Sampling, row, col, ROW, COL, channels);
		break;
	case Iterpolation::BICUBIC:
		Bicubic_Kernel_Y <<<gridDim, blockDim>>>(dLR, dRI, Sampling, row, col, ROW, COL, channels);
		hipDeviceSynchronize();
		Bicubic_Kernel_X <<<gridDim, blockDim >>>(dLR, dRI, Sampling, row, col, ROW, COL, channels);
		break;
	case Iterpolation::DEFAULT:
		Nearest_Neighborhood_Kernel << <gridDim, blockDim >> >(dLR, dRI, Sampling, row, col, ROW, COL, channels);
		break;
	}
	hipDeviceSynchronize();
	timer->offTimer(timerID);

	hipMemcpy(RI.data, dRI, RI_Size, hipMemcpyDeviceToHost);

	/*
	printf("GPU\n");
	for (int c = 0; c < channels; c++) {
		for (int y = 0; y < RI.rows; y++) {
			for (int x = 0; x < RI.cols; x++) {
				printf("%3d ", (int)RI.at<Vec3b>(y, x)[c]);
			}
			printf("\n");
		}
		printf("\n");
	}*/

	hipFree(dLR);
	hipFree(dRI);
	return RI;
}

Mat Nearest_Neighborhood(Mat LR, int Sampling) { //�ִ����� ������
	int row = LR.rows;
	int col = LR.cols;
	int channels = LR.channels();

	int ROW = row * Sampling - (Sampling - 1);
	int COL = col * Sampling - (Sampling - 1);
	Mat RI = Mat::zeros(ROW, COL, LR.type());

	
	for (int RIy = 0; RIy < ROW; RIy++) {
		for (int RIx = 0; RIx < COL; RIx++) {
			int LRy = floor((float)RIy / Sampling + 0.5);
			int LRx = floor((float)RIx / Sampling + 0.5);
			RI.at<Vec3b>(RIy, RIx) = LR.at<Vec3b>(LRy, LRx);
		}
	}
	return RI;
}

Mat Bilinear(Mat LR, int Sampling) {  //����������
	int row = LR.rows;
	int col = LR.cols;
	int channels = LR.channels();

	int ROW = row * Sampling - (Sampling - 1);
	int COL = col * Sampling - (Sampling - 1);

	Mat RI = Mat::zeros(ROW, COL, LR.type());

	
	for (int RIy = 0; RIy < ROW; RIy++) {
		for (int RIx = 0; RIx < COL; RIx++) {
			for (int c = 0; c < channels; c++) {
				int LRy = (float)RIy / Sampling;
				int LRx = (float)RIx / Sampling;
				int m = RIy % Sampling;
				int n = RIx % Sampling;
				if (m == 0 && n == 0) {
					RI.at<Vec3b>(RIy, RIx) = LR.at<Vec3b>(LRy, LRx);
					break;
				}
				else {
					if (RIy == ROW - 1) {
						LRy--;
						m = Sampling - m;
					}
					if (RIx == COL - 1) {
						LRx--;
						n = Sampling - n;
					}

					PIXEL v1 = LR.at<Vec3b>((LRy + 0), (LRx + 0))[c];
					PIXEL v2 = LR.at<Vec3b>((LRy + 0), (LRx + 1))[c];
					PIXEL v3 = LR.at<Vec3b>((LRy + 1), (LRx + 0))[c];
					PIXEL v4 = LR.at<Vec3b>((LRy + 1), (LRx + 1))[c];

					double w1 = (double)((Sampling - m) * (Sampling - n)) / (Sampling * Sampling);
					double w2 = (double)((Sampling - m) * (n)) / (Sampling * Sampling);
					double w3 = (double)((m) * (Sampling - n)) / (Sampling * Sampling);
					double w4 = (double)((m) * (n)) / (Sampling * Sampling);
					RI.at<Vec3b>(RIy, RIx)[c] = ((v1 * w1) + (v2 * w2) + (v3 * w3) + (v4 * w4));
				}
			}
		}
	}
	return RI;
}

Mat Bicubic(Mat LR, int Sampling) {
	int row = LR.rows;
	int col = LR.cols;
	int channels = LR.channels();

	int ROW = row * Sampling - (Sampling - 1);
	int COL = col * Sampling - (Sampling - 1);

	Mat RI = Mat::zeros(ROW, COL, LR.type());
	
	for (int RIy = 0; RIy < ROW; RIy++) {
		for (int RIx = 0; RIx < COL; RIx += Sampling) {
			for (int c = 0; c < channels; c++) {
				int LRy = (float)RIy / Sampling;
				int LRx = (float)RIx / Sampling;
				int m = RIy % Sampling;
				int n = RIx % Sampling;
				if (m == 0 && n == 0) {
					RI.at<Vec3b>(RIy, RIx) = LR.at<Vec3b>(LRy, LRx);
					break;
				}

				double d = (double)m / Sampling;
				if (LRy == 0) {
					LRy += 1; d -= 1;
				}
				else if (LRy == row - 2) {
					LRy -= 1; d += 1;
				}
				PIXEL v1 = LR.at<Vec3b>(LRy - 1, LRx)[c];
				PIXEL v2 = LR.at<Vec3b>(LRy + 0, LRx)[c];
				PIXEL v3 = LR.at<Vec3b>(LRy + 1, LRx)[c];
				PIXEL v4 = LR.at<Vec3b>(LRy + 2, LRx)[c];
				
				RI.at<Vec3b>(RIy, RIx)[c] = cubic(v1, v2, v3, v4, d);
			}
		}
	}

	for (int RIy = 0; RIy < ROW; RIy++) {
		for (int RIx = 1; RIx < COL; RIx++) {
			for (int c = 0; c < channels; c++) {
				int n = RIx % Sampling;
				int tRIx = RIx;
				if (n == 0) break;

				double d = (double)n / Sampling;
				if (tRIx <= Sampling) {
					tRIx += Sampling; d -= 1;
				}
				else if (tRIx >= COL - Sampling) {
					tRIx -= Sampling; d += 1;
				}
				PIXEL v1 = RI.at<Vec3b>(RIy, (tRIx - n) - (1 * Sampling))[c];
				PIXEL v2 = RI.at<Vec3b>(RIy, (tRIx - n) + (0 * Sampling))[c];
				PIXEL v3 = RI.at<Vec3b>(RIy, (tRIx - n) + (1 * Sampling))[c];
				PIXEL v4 = RI.at<Vec3b>(RIy, (tRIx - n) + (2 * Sampling))[c];

				RI.at<Vec3b>(RIy, RIx)[c] = cubic(v1, v2, v3, v4, d);
			}
		}
	}
	return RI;
}

Mat CPU_Call(Mat LR, int Sampling, int mod) {
	Mat result;
	switch (mod) {
	case Iterpolation::NN:
		result = Nearest_Neighborhood(LR, Sampling);
		break;
	case Iterpolation::BILINEAR:
		result = Bilinear(LR, Sampling);
		break;
	case Iterpolation::BICUBIC:
		result = Bicubic(LR, Sampling);
		break;
	case Iterpolation::DEFAULT:
		result = Nearest_Neighborhood(LR, Sampling);
		break;
	}
	return result;
}